//////////////////////////////////////////////////////////////////////
// A simple example to show how CUDA WMMA API works with Tensor Cores
//    Created by Zong-Sheng Wang @ 2018/11/25
// Performance Tips:
//    To minimize bank conflicts, you should try to shift row or 
// column of matrics in shared memory
// cmd: 
//    $ nvcc -o main main.cu -arch sm_75

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <mma.h>



// NVML
#include <nvml.h>

#define WARP_SIZE 32

// MMA matrix tile dimensions.
#define M 8
#define N 8
#define K 4

// GEMM configuration.
#define M_TILES 512
#define N_TILES 512
#define K_TILES 512

#define M_TOTAL (M * M_TILES)
#define N_TOTAL (N * N_TILES)
#define K_TOTAL (K * K_TILES)


//__global__ void WMMAINT8()
using namespace nvcuda;

__host__ void InitMatrix(double *A, double *B, double *C)
{
  for (int i = 0; i < M_TOTAL*K_TOTAL; i++)
    A[i] = (rand() % 1000 / 1000.0f);
  for (int i = 0; i < K_TOTAL*N_TOTAL; i++)
    B[i] = (rand() % 1000 / 1000.0f);
  for (int i = 0; i < M_TOTAL*N_TOTAL; i++)
    C[i] = rand() % 1000 / 1000.0f;
}



__global__ void WMMAF16TensorCore(double *A, double *B, double *C, double *D)
{
  int ix = (blockIdx.x * blockDim.x + threadIdx.x)/WARP_SIZE;
  int iy = (blockIdx.y * blockDim.y + threadIdx.y);
  
  wmma::fragment<wmma::matrix_a, M, N, K, double, wmma::row_major> a_frag;
  wmma::fragment<wmma::matrix_b, M, N, K, double, wmma::col_major> b_frag;
  wmma::fragment<wmma::accumulator, M, N, K, double> ab_frag;
  wmma::fragment<wmma::accumulator, M, N, K, double> c_frag;
  
  wmma::fill_fragment(ab_frag, 0.0f);


  // AB = A*B
  int a_col, a_row, b_col, b_row, c_col, c_row;
  a_row = ix * M;
  b_row = iy * N;

  wmma::load_matrix_sync(a_frag, A + 0 + a_row * M_TOTAL, M_TOTAL);
  wmma::load_matrix_sync(b_frag, B + 0 + 0 * K_TOTAL, K_TOTAL);
  wmma::load_matrix_sync(c_frag, C + c_col + c_row * N_TOTAL, N_TOTAL, wmma::mem_row_major);
  for (int k=0; k<K_TOTAL; k+=K) {
    a_col = b_col = k;

    if (a_row < M_TOTAL && a_col < K_TOTAL && b_row < K_TOTAL && b_col < N_TOTAL) {
      // Load the inputs
      // wmma::load_matrix_sync(a_frag, A + a_col + a_row * M_TOTAL, M_TOTAL);
      // wmma::load_matrix_sync(b_frag, B + b_col + b_col * K_TOTAL, K_TOTAL);

      // Perform the matrix multiplication
      wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }
    // wmma::store_matrix_sync(D + c_col + c_row * N_TOTAL, c_frag, N_TOTAL, wmma::mem_row_major);
  }

  // D = AB + C
  c_col = b_row;
  c_row = a_row;
  if (c_row < M_TOTAL && c_col < N_TOTAL) {
    // wmma::load_matrix_sync(c_frag, C + c_col + c_row * N_TOTAL, N_TOTAL, wmma::mem_row_major);

    // for (int i = 0; i < c_frag.num_elements; i++) {
    //   c_frag.x[i] = ab_frag.x[i] + c_frag.x[i];
    // }

    // Store the output
    wmma::store_matrix_sync(D + c_col + c_row * N_TOTAL, c_frag, N_TOTAL, wmma::mem_row_major);
  }
}

hipError_t CalcWMMA(double *A, double *B, double *C, double *D)
{
  hipError_t cuda_status;
  dim3 gridDim, blockDim;
  // 16 warps in one block
  blockDim.x = 4 * WARP_SIZE; 
  blockDim.y = 4;

  gridDim.x = (M_TOTAL + (M * blockDim.x / WARP_SIZE - 1)) / (M * blockDim.x / WARP_SIZE);
  gridDim.y = (N_TOTAL + N * blockDim.y - 1) / (N * blockDim.y);
  int repeat=500;
  // for Performance Metrics
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
   unsigned int power1, power2;
    nvmlReturn_t result;
  nvmlDevice_t device;
  nvmlEnableState_t mode;

    result=nvmlInit();
  result = nvmlDeviceGetHandleByIndex(0, &device);
  // assert(NVML_SUCCESS == result);
  result=nvmlDeviceGetPowerManagementMode(device, &mode);
  // printf("enabled = %d\n", mode);
  result=nvmlDeviceGetPowerUsage(device,&power1);
  // assert(NVML_SUCCESS == result);
  for(int i=0; i<repeat; i++)
  {
    WMMAF16TensorCore<<<gridDim, blockDim>>>(A, B, C, D);
  }
  hipEventRecord(start);
  for(int i=0; i<repeat; i++)
  {
    WMMAF16TensorCore<<<gridDim, blockDim>>>(A, B, C, D);
  }
  
  
  
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  cuda_status = hipDeviceSynchronize();
  result=nvmlDeviceGetPowerUsage(device,&power2);
  // assert(NVML_SUCCESS == result);
  nvmlShutdown();

  float milliseconds = 0;

  hipEventElapsedTime(&milliseconds, start, stop);

  // for Performance Metrics
  printf("[+] GPU(with Tensor Cores) Elapsed Time: %f ms\n", milliseconds);
  // references from https://devblogs.nvidia.com/how-implement-performance-metrics-cuda-cc/
  printf("[+] TFLOPS: %.2f\n", ((double)M_TOTAL * N_TOTAL* K_TOTAL * 2)*repeat / milliseconds / 1e9);
  printf("power from %u W to %u W\n", 
                      power1/1000, power2/1000);
    // printf("%f, ", gflops);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return cuda_status;
}


int main()
{
  hipError_t cuda_status;
  cuda_status = hipSetDevice(0);
  if (cuda_status != hipSuccess) {
    printf("hipSetDevice failed! ");
    return 1;
  }


  // Matrix on device
  double *A;
  double *B;
  double *C;
  double *D;

  // CUDA Unified Memory 
  hipMallocManaged((void **)&A, sizeof(double) * M_TOTAL * K_TOTAL);
  hipMallocManaged((void **)&B, sizeof(double) * K_TOTAL * N_TOTAL);
  hipMallocManaged((void **)&C, sizeof(double) * M_TOTAL * N_TOTAL);
  hipMallocManaged((void **)&D, sizeof(double) * M_TOTAL * N_TOTAL);
  
  // Init matrix A B C on host
  //InitHostMatrix(host_A, host_B, host_C);
  printf("[*] Initializing Matrix...\n");
  InitMatrix(A, B, C);
  printf("[+]   A: %d x %d\n", M_TOTAL, K_TOTAL);
  printf("[+]   B: %d x %d\n", K_TOTAL, N_TOTAL);
  printf("[+]   C: %d x %d\n", M_TOTAL, N_TOTAL);
  
  // computing gemm using tensor core
  printf("[*] Computing D = A * B +C with Tensor Cores...\n");
  // D = A * B +C, D holds the result after ret
  cuda_status = CalcWMMA(A, B, C, D);
  
  cuda_status = hipDeviceReset();
  if (cuda_status != hipSuccess) {
    printf("hipDeviceReset failed! ");
    return 1;
  }
  // Todo: Add a function to verify the result by using the result of CPU version implementation.

  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(D);

  return 0;
}