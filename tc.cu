//////////////////////////////////////////////////////////////////////
// A simple example to show how CUDA WMMA API works with Tensor Cores
//    Created by Zong-Sheng Wang @ 2018/11/25
// Performance Tips:
//    To minimize bank conflicts, you should try to shift row or 
// column of matrics in shared memory
// cmd: 
//    $ nvcc -o main main.cu -arch sm_75

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <mma.h>



// NVML
#include <nvml.h>

#define WARP_SIZE 32

// MMA matrix tile dimensions.
#define M 16
#define N 16
#define K 8

// GEMM configuration.
#define M_TILES 512
#define N_TILES 512
#define K_TILES 512

#define M_TOTAL (M * M_TILES)
#define N_TOTAL (N * N_TILES)
#define K_TOTAL (K * K_TILES)


//__global__ void WMMAINT8()
using namespace nvcuda;

__host__ void InitMatrix(float *A, float *B, float *C)
{
  for (int i = 0; i < M_TOTAL*K_TOTAL; i++)
    A[i] = (rand() % 1000 / 1000.0f);
  for (int i = 0; i < K_TOTAL*N_TOTAL; i++)
    B[i] = (rand() % 1000 / 1000.0f);
  for (int i = 0; i < M_TOTAL*N_TOTAL; i++)
    C[i] = rand() % 1000 / 1000.0f;
}



__global__ void WMMAF16TensorCore(float *A, float *B, float *C, float *D)
{
  int ix = (blockIdx.x * blockDim.x + threadIdx.x)/WARP_SIZE;
  int iy = (blockIdx.y * blockDim.y + threadIdx.y);
  
  wmma::fragment<wmma::matrix_a, M, N, K, wmma::precision::tf32, wmma::row_major> a0_frag;
  wmma::fragment<wmma::matrix_a, M, N, K, wmma::precision::tf32, wmma::row_major> a1_frag;
  wmma::fragment<wmma::matrix_a, M, N, K, wmma::precision::tf32, wmma::row_major> a2_frag;
  wmma::fragment<wmma::matrix_a, M, N, K, wmma::precision::tf32, wmma::row_major> a3_frag;
  wmma::fragment<wmma::matrix_b, M, N, K, wmma::precision::tf32, wmma::col_major> b0_frag;
  wmma::fragment<wmma::matrix_b, M, N, K, wmma::precision::tf32, wmma::col_major> b1_frag;
  wmma::fragment<wmma::matrix_b, M, N, K, wmma::precision::tf32, wmma::col_major> b2_frag;
  wmma::fragment<wmma::matrix_b, M, N, K, wmma::precision::tf32, wmma::col_major> b3_frag;
  wmma::fragment<wmma::accumulator, M, N, K, float> ab_frag;
  wmma::fragment<wmma::accumulator, M, N, K, float> c_frag;
  wmma::fragment<wmma::accumulator, M, N, K, float> c0_frag;
  wmma::fragment<wmma::accumulator, M, N, K, float> c1_frag;
  wmma::fragment<wmma::accumulator, M, N, K, float> c2_frag;
  wmma::fragment<wmma::accumulator, M, N, K, float> c3_frag;
  
  wmma::fill_fragment(ab_frag, 0.0f);


  // AB = A*B
  int a_col, a_row, b_col, b_row, c_col, c_row;
  a_row = ix * M;
  b_row = iy * N;

  wmma::load_matrix_sync(a0_frag, A + 0 + a_row * M_TOTAL, M_TOTAL);
  wmma::load_matrix_sync(a1_frag, A + 1 + a_row * M_TOTAL, M_TOTAL);
  wmma::load_matrix_sync(a2_frag, A + 2 + a_row * M_TOTAL, M_TOTAL);
  wmma::load_matrix_sync(a3_frag, A + 3 + a_row * M_TOTAL, M_TOTAL);
  for (int t = 0; t < a0_frag.num_elements; t++) {
        a0_frag.x[t] = wmma::__float_to_tf32(a0_frag.x[t]);
        a1_frag.x[t] = wmma::__float_to_tf32(a1_frag.x[t]);
        a2_frag.x[t] = wmma::__float_to_tf32(a2_frag.x[t]);
        a3_frag.x[t] = wmma::__float_to_tf32(a3_frag.x[t]);
    }

  wmma::load_matrix_sync(b0_frag, B + 0 + 0 * K_TOTAL, K_TOTAL);
  wmma::load_matrix_sync(b1_frag, B + 0 + 0 * K_TOTAL, K_TOTAL);
  wmma::load_matrix_sync(b2_frag, B + 0 + 0 * K_TOTAL, K_TOTAL);
  wmma::load_matrix_sync(b3_frag, B + 0 + 0 * K_TOTAL, K_TOTAL);
for (int t = 0; t < b0_frag.num_elements; t++) {
        b0_frag.x[t] = wmma::__float_to_tf32(b0_frag.x[t]);
        b1_frag.x[t] = wmma::__float_to_tf32(b1_frag.x[t]);
        b2_frag.x[t] = wmma::__float_to_tf32(b2_frag.x[t]);
        b3_frag.x[t] = wmma::__float_to_tf32(b3_frag.x[t]);
    }
  wmma::load_matrix_sync(c_frag, C + c_col + c_row * N_TOTAL, N_TOTAL, wmma::mem_row_major);
  wmma::load_matrix_sync(c0_frag, C + c_col + c_row * N_TOTAL, N_TOTAL, wmma::mem_row_major);
  wmma::load_matrix_sync(c1_frag, C + c_col + c_row * N_TOTAL, N_TOTAL, wmma::mem_row_major);
  wmma::load_matrix_sync(c2_frag, C + c_col + c_row * N_TOTAL, N_TOTAL, wmma::mem_row_major);
  wmma::load_matrix_sync(c3_frag, C + c_col + c_row * N_TOTAL, N_TOTAL, wmma::mem_row_major);
  for (int k=0; k<K_TOTAL; k+=K) {
    a_col = b_col = k;

    if (a_row < M_TOTAL && a_col < K_TOTAL && b_row < K_TOTAL && b_col < N_TOTAL) {
      // Load the inputs
      // wmma::load_matrix_sync(a_frag, A + a_col + a_row * M_TOTAL, M_TOTAL);
      // wmma::load_matrix_sync(b_frag, B + b_col + b_col * K_TOTAL, K_TOTAL);

      // Perform the matrix multiplication
      // wmma::mma_sync(ab_frag, a_frag, b_frag, ab_frag);
      wmma::mma_sync(c0_frag, a0_frag, b0_frag, c0_frag);
      wmma::mma_sync(c1_frag, a1_frag, b1_frag, c1_frag);
      wmma::mma_sync(c2_frag, a2_frag, b2_frag, c2_frag);
      wmma::mma_sync(c3_frag, a3_frag, b3_frag, c3_frag);
    }
    // wmma::store_matrix_sync(D + c_col + c_row * N_TOTAL, c_frag, N_TOTAL, wmma::mem_row_major);
  }

  // D = AB + C
  c_col = b_row;
  c_row = a_row;
  if (c_row < M_TOTAL && c_col < N_TOTAL) {
    // wmma::load_matrix_sync(c_frag, C + c_col + c_row * N_TOTAL, N_TOTAL, wmma::mem_row_major);

    for (int i = 0; i < c_frag.num_elements; i++) {
      c_frag.x[i] = c0_frag.x[i] + c1_frag.x[i]+c2_frag.x[i]+c3_frag.x[i];;
    }

    // Store the output
    wmma::store_matrix_sync(D + c_col + c_row * N_TOTAL, c_frag, N_TOTAL, wmma::mem_row_major);
  }
}

hipError_t CalcWMMA(float *A, float *B, float *C, float *D)
{
  hipError_t cuda_status;
  dim3 gridDim, blockDim;
  // 16 warps in one block
  blockDim.x = 4 * WARP_SIZE; 
  blockDim.y = 4;

  gridDim.x = (M_TOTAL + (M * blockDim.x / WARP_SIZE - 1)) / (M * blockDim.x / WARP_SIZE);
  gridDim.y = (N_TOTAL + N * blockDim.y - 1) / (N * blockDim.y);
  int repeat=500;
  // for Performance Metrics
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
   unsigned int power1, power2;
    nvmlReturn_t result;
  nvmlDevice_t device;
  nvmlEnableState_t mode;

    result=nvmlInit();
  result = nvmlDeviceGetHandleByIndex(0, &device);
  // assert(NVML_SUCCESS == result);
  result=nvmlDeviceGetPowerManagementMode(device, &mode);
  // printf("enabled = %d\n", mode);
  result=nvmlDeviceGetPowerUsage(device,&power1);
  // assert(NVML_SUCCESS == result);
  for(int i=0; i<repeat; i++)
  {
    WMMAF16TensorCore<<<gridDim, blockDim>>>(A, B, C, D);
  }
  hipEventRecord(start);
  for(int i=0; i<repeat; i++)
  {
    WMMAF16TensorCore<<<gridDim, blockDim>>>(A, B, C, D);
  }
  
  
  
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  cuda_status = hipDeviceSynchronize();
  result=nvmlDeviceGetPowerUsage(device,&power2);
  // assert(NVML_SUCCESS == result);
  nvmlShutdown();

  float milliseconds = 0;

  hipEventElapsedTime(&milliseconds, start, stop);

  // for Performance Metrics
  printf("[+] GPU(with Tensor Cores) Elapsed Time: %f ms\n", milliseconds);
  // references from https://devblogs.nvidia.com/how-implement-performance-metrics-cuda-cc/
  printf("[+] TFLOPS: %.2f\n", ((float)M_TOTAL *4* N_TOTAL* K_TOTAL * 2)*repeat / milliseconds / 1e9);
  printf("power from %u W to %u W\n", 
                      power1/1000, power2/1000);
    // printf("%f, ", gflops);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return cuda_status;
}


int main()
{
  hipError_t cuda_status;
  cuda_status = hipSetDevice(0);
  if (cuda_status != hipSuccess) {
    printf("hipSetDevice failed! ");
    return 1;
  }


  // Matrix on device
  float *A;
  float *B;
  float *C;
  float *D;

  // CUDA Unified Memory 
  hipMallocManaged((void **)&A, sizeof(float) * M_TOTAL * K_TOTAL);
  hipMallocManaged((void **)&B, sizeof(float) * K_TOTAL * N_TOTAL);
  hipMallocManaged((void **)&C, sizeof(float) * M_TOTAL * N_TOTAL);
  hipMallocManaged((void **)&D, sizeof(float) * M_TOTAL * N_TOTAL);
  
  // Init matrix A B C on host
  //InitHostMatrix(host_A, host_B, host_C);
  printf("[*] Initializing Matrix...\n");
  InitMatrix(A, B, C);
  printf("[+]   A: %d x %d\n", M_TOTAL, K_TOTAL);
  printf("[+]   B: %d x %d\n", K_TOTAL, N_TOTAL);
  printf("[+]   C: %d x %d\n", M_TOTAL, N_TOTAL);
  
  // computing gemm using tensor core
  printf("[*] Computing D = A * B +C with Tensor Cores...\n");
  // D = A * B +C, D holds the result after ret
  cuda_status = CalcWMMA(A, B, C, D);
  
  cuda_status = hipDeviceReset();
  if (cuda_status != hipSuccess) {
    printf("hipDeviceReset failed! ");
    return 1;
  }
  // Todo: Add a function to verify the result by using the result of CPU version implementation.

  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(D);

  return 0;
}