#include "hip/hip_runtime.h"
#include <assert.h>
#include <iostream>
#include "stdio.h"
// #include "launchHelper.cuh"
// // CBLAS (OpenBLAS)
// #include "cblas.h"

// CUDA runtime
#include <hip/hip_runtime.h>
#include <>
#include "omp.h"
// CUBLAS
#include <hipblas.h>

// NVML
#include <nvml.h>
#include<unistd.h>   
/**
 * Panic wrapper for unwinding CUDA runtime errors
 */
#define CUDA_CHECK(status)                                                  \
    {                                                                       \
        hipError_t error = status;                                         \
        if (error != hipSuccess) {                                         \
            printf ("ERROR : %s %d CUDA : %s\n", __FILE__,  __LINE__, hipGetErrorString(error));           \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    }

/**
 * Panic wrapper for unwinding CUBLAS runtime errors
 */
#define CUBLAS_CHECK(status)                                        \
    {                                                     \
        hipblasStatus_t error = status;                                  \
        if(error != HIPBLAS_STATUS_SUCCESS) {                        \
            printf ("ERROR : %s %d CUBLAS", __FILE__,  __LINE__);       \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    }


typedef double typ;

void init(typ *buf, int size) {
    for (int i = 0; i < size; ++i) {
        buf[i] = (typ)1.0f * rand() / RAND_MAX;
        //buf[i] = 1.0f;
    }
}

static float peak_flops;
static float get_peak_flops(){
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

  int Clock = prop.clockRate;
  int SM_nums = prop.multiProcessorCount;
    // int FP_units_pre_SM = 4 * 16 * 2; // RTX 3090 w/o tensor core
    int FP_units_pre_SM = 4 * 16; // A100 w/o tensor core
  float gflops = 2.0 * Clock * SM_nums * FP_units_pre_SM / 1e6;

    printf("Name:\t%s\n", prop.name);
    printf("Clock rate:\t%d\n", Clock);
    printf("Multiprocessor count:\t%d\n", SM_nums);
    printf("FP units count pre SM:\t%d\n", FP_units_pre_SM);
  printf("peakGFLOPs: %f\n", gflops);
  return gflops;
}

void becnmark_cublas(int M, int N, int K, int n_loops) {
    const int lda = K;
    const int ldb = N;
    const int ldc = N;
    const int a_alloc = sizeof(typ) * M * lda;
    const int b_alloc = sizeof(typ) * K * ldb;
    const int c_alloc = sizeof(typ) * M * ldc;

    typ* h_A =    (typ*)malloc(a_alloc);
    typ* h_B =    (typ*)malloc(b_alloc);
    typ* h_C =    (typ*)malloc(c_alloc);
    typ* h_refC = (typ*)malloc(c_alloc);
    typ alpha = 1.0;
    typ beta = 0.0;

    init(h_A, M * lda);
    init(h_B, K * ldb);

    typ* d_A;
    typ* d_B;
    typ* d_C;

    CUDA_CHECK(hipMalloc(&d_A, a_alloc));
    CUDA_CHECK(hipMalloc(&d_B, b_alloc));
    CUDA_CHECK(hipMalloc(&d_C, c_alloc));
    
    CUDA_CHECK(hipMemcpy( d_A, h_A, a_alloc, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy( d_B, h_B, b_alloc, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset( d_C, 0.f, c_alloc));

    // Initialization power
    unsigned int power1;
    nvmlReturn_t result;
  nvmlDevice_t device;
  nvmlEnableState_t mode;

    result=nvmlInit();
  result = nvmlDeviceGetHandleByIndex(0, &device);
  assert(NVML_SUCCESS == result);
  result=nvmlDeviceGetPowerManagementMode(device, &mode);

  result=nvmlDeviceGetPowerUsage(device,&power1);
  assert(NVML_SUCCESS == result);
  hipDeviceSynchronize();

    // Initialization timing
    
    // CUBLAS_CHECK(hipblasSetMathMode( blas_handle, HIPBLAS_TENSOR_OP_MATH ));
    // CUDA_CHECK(hipMemcpy( d_C, h_refC, c_alloc, hipMemcpyHostToDevice));
  #pragma omp parallel num_threads(2)
  {
    if (omp_get_thread_num() == 1)
    {
      for(int i=0; i<10;i++)
      {
        unsigned int power1;
        unsigned int clock;
        result=nvmlDeviceGetPowerUsage(device,&power1);
        result=nvmlDeviceGetClock(device,NVML_CLOCK_SM,NVML_CLOCK_ID_CURRENT,&clock);
        // cuda_status = hipDeviceSynchronize();
        // hipDeviceProp_t prop;
        // hipGetDeviceProperties ( &prop, 0 );
        assert(NVML_SUCCESS == result);
        printf("%d power  %u W in requency %d MHz\n", i,
                        power1/1000, clock);
        sleep(1);
      }
    }
    if (omp_get_thread_num() == 0){
        hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));
        float msecTotal = 0;

        // cublas
        hipblasHandle_t blas_handle;  
        CUBLAS_CHECK(hipblasCreate(&blas_handle));
        for (int run = 0 ; run < n_loops; run ++ ) {
            CUBLAS_CHECK(
                hipblasDgemm (blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    N, M, K, &alpha, 
                    d_B, ldb, d_A, lda, &beta, d_C, ldc
                )
            );
        }
        
        CUDA_CHECK(hipEventRecord(start));
        for (int run = 0 ; run < n_loops; run ++ ) {
            CUBLAS_CHECK(
                hipblasDgemm (blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    N, M, K, &alpha, 
                    d_B, ldb, d_A, lda, &beta, d_C, ldc
                )
            );
        }
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK(hipEventElapsedTime(&msecTotal, start, stop));
        float latency = msecTotal;
        float tflops = 2.0 * M * N * K / latency / 1e6 * n_loops/1000;
        printf("CUBLAS, M: %d, N: %d, K: %d, perf: %.2f tflops,  latency: %.6f ms\n", 
                          M, N, K, tflops, latency / n_loops);
        CUBLAS_CHECK(hipblasDestroy(blas_handle)); 
    }
  }
    hipDeviceSynchronize();

    CUDA_CHECK(hipMemcpy( h_C, d_C, c_alloc, hipMemcpyDeviceToHost));
    


    // Free Memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_refC);
}

int main(void) {

    int m = 10240;
    int n = 10240;
    int k = 2048;
    int repeats = 100;

    peak_flops = get_peak_flops();
    becnmark_cublas(m, n, k, repeats);

  return 0;
}
