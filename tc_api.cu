#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////
// A simple example to show how CUDA WMMA API works with Tensor Cores
//    Created by Zong-Sheng Wang @ 2018/11/25
// Performance Tips:
//    To minimize bank conflicts, you should try to shift row or 
// column of matrics in shared memory
// cmd: 
//    $ nvcc -o main main.cu -arch sm_75

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <mma.h>

#include "hip/hip_runtime.h"
#include ""
// NVML
#include <nvml.h>

#define WARP_SIZE 32

// MMA matrix tile dimensions.
#define M 16
#define N 16
#define K 8

// GEMM configuration.
#define M_TILES 512
#define N_TILES 512
#define K_TILES 512
#define TILE 4
// #define M_TOTAL (M * M_TILES)
// #define N_TOTAL (N * N_TILES)
// #define K_TOTAL (K * K_TILES)
#define ILP 4

//__global__ void WMMAINT8()
using namespace nvcuda;
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
 }                                                                 \
}


__host__ void InitMatrix(float *A, float *B, float *C, int tiles)
{
  int M_TOTAL=(M );
  int N_TOTAL=(N );
  int K_TOTAL=(K );

  for (int i = 0; i < M_TOTAL*K_TOTAL; i++)
    A[i] = (rand() % 1000 / 1000.0f);
  for (int i = 0; i < K_TOTAL*N_TOTAL; i++)
    B[i] = (rand() % 1000 / 1000.0f);
  for (int i = 0; i < M_TOTAL*N_TOTAL; i++)
    C[i] = rand() % 1000 / 1000.0f;
}



__global__ void WMMAF16TensorCore(float *A, float *B, float *C, float *D, int tiles)
{
  int ix = (blockIdx.x * blockDim.x + threadIdx.x)/WARP_SIZE;
  int iy = (blockIdx.y * blockDim.y + threadIdx.y);
  
  int M_TOTAL=(M * tiles );
  int N_TOTAL=(N * tiles );
  int K_TOTAL=(K * tiles );

  wmma::fragment<wmma::matrix_a, M, N, K, wmma::precision::tf32, wmma::row_major> a_frag[ILP];
  wmma::fragment<wmma::matrix_b, M, N, K, wmma::precision::tf32, wmma::col_major> b_frag[ILP];
  wmma::fragment<wmma::accumulator, M, N, K, float> c_frag[ILP];
  #pragma unroll
  for(int ilp=0; ilp<ILP; ilp++)
  {
    wmma::fill_fragment(c_frag[ilp], 0.0f);
  }
  

  // AB = A*B 
  int a_col, a_row, b_col, b_row, c_col, c_row;
  a_row = ix * M;
  b_row = iy * N;

  // wmma::load_matrix_sync(a_frag, A + 0 + a_row * M_TOTAL, M_TOTAL);
  #pragma unroll
  for(int ilp=0; ilp<ILP; ilp++)
  {
    wmma::load_matrix_sync(a_frag[ilp], A + 0 + 0 * M_TOTAL, M_TOTAL);
    for (int t = 0; t < a_frag[ilp].num_elements; t++) {
        a_frag[ilp].x[t] = wmma::__float_to_tf32(a_frag[ilp].x[t]);
    }
  }
  
  #pragma unroll
  for(int ilp=0; ilp<ILP; ilp++)
  {
    wmma::load_matrix_sync(b_frag[ilp], B + 0 + 0 * K_TOTAL, K_TOTAL);
    for (int t = 0; t < b_frag[ilp].num_elements; t++) {
          b_frag[ilp].x[t] = wmma::__float_to_tf32(b_frag[ilp].x[t]);
    }
  }

  #pragma unroll
  for(int ilp=0; ilp<ILP; ilp++)
  {
    for (int k=0; k<K_TOTAL; k+=K) {
      a_col = b_col = k;
      {
        wmma::mma_sync(c_frag[ilp], a_frag[ilp], b_frag[ilp], c_frag[ilp]);
      }
    }
  }

  c_col = b_row;
  c_row = a_row;
  // if (c_row < M_TOTAL && c_col < N_TOTAL) 
  #pragma unroll
  for(int ilp=0; ilp<ILP; ilp++)
  {
    for (int i = 0; i < c_frag[ilp].num_elements; i++) {
      c_frag[ilp].x[i] = c_frag[ilp].x[i];// + c1_frag.x[i]+c2_frag.x[i]+c3_frag.x[i];
    }
    wmma::store_matrix_sync(D + c_col + c_row * N_TOTAL, c_frag[ilp], N_TOTAL, wmma::mem_row_major);
  }
}

hipError_t CalcWMMA(float *A, float *B, float *C, float *D, int tiles)
{
  int M_TOTAL=(M * tiles );
  int N_TOTAL=(N * tiles );
  int K_TOTAL=(K * tiles );

  hipError_t cuda_status;
  dim3 gridDim, blockDim;
  // 16 warps in one block

  
  blockDim.x = 4 * WARP_SIZE; 
  blockDim.y = 4;

  gridDim.x = (M_TOTAL + (M * blockDim.x / WARP_SIZE - 1)) / (M * blockDim.x / WARP_SIZE);
  gridDim.y = (N_TOTAL + N * blockDim.y - 1) / (N * blockDim.y);


  int repeat=1000;
  // for Performance Metrics
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  unsigned int power1, power2;
  // nvmlReturn_t result;
  nvmlDevice_t device;
  nvmlEnableState_t mode;

  result=nvmlInit();
  result = nvmlDeviceGetHandleByIndex(0, &device);
  // assert(NVML_SUCCESS == result);
  result=nvmlDeviceGetPowerManagementMode(device, &mode);
  // printf("enabled = %d\n", mode);
  result=nvmlDeviceGetPowerUsage(device,&power1);
  // assert(NVML_SUCCESS == result);
  for(int i=0; i<repeat; i++)
  {
    WMMAF16TensorCore<<<gridDim, blockDim>>>(A, B, C, D, tiles);
  }
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventRecord(start);
  for(int i=0; i<repeat; i++)
  {
    WMMAF16TensorCore<<<gridDim, blockDim>>>(A, B, C, D, tiles);
  }
  
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  cuda_status = hipDeviceSynchronize();
  result=nvmlDeviceGetPowerUsage(device,&power2);
  // assert(NVML_SUCCESS == result);
  nvmlShutdown();

  float milliseconds = 0;

  hipEventElapsedTime(&milliseconds, start, stop);

  // for Performance Metrics
  printf("[+] GPU(with Tensor Cores) Elapsed Time: %f ms\n", milliseconds);
  // references from https://devblogs.nvidia.com/how-implement-performance-metrics-cuda-cc/
  printf("[+] TFLOPS: %.2f\n", ((float)M_TOTAL * ILP * N_TOTAL* K_TOTAL * 2)*repeat / milliseconds / 1e9);
  printf("power from %u W to %u W\n", 
                      power1/1000, power2/1000);
    // printf("%f, ", gflops);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return cuda_status;
}


int main()
{
  hipError_t cuda_status;
  cuda_status = hipSetDevice(0);
  if (cuda_status != hipSuccess) {
    printf("hipSetDevice failed! ");
    return 1;
  }
  int sm_count;
  hipDeviceGetAttribute ( &sm_count, hipDeviceAttributeMultiprocessorCount,0 );
  int tiles=sm_count * TILE;
  int M_TOTAL=(M * tiles);
  int N_TOTAL=(N * tiles);
  int K_TOTAL=(K * tiles);

  // Matrix on device
  float *A;
  float *B;
  float *C;
  float *D;

  // CUDA Unified Memory 
  hipMallocManaged((void **)&A, sizeof(float) * M_TOTAL * K_TOTAL);
  hipMallocManaged((void **)&B, sizeof(float) * K_TOTAL * N_TOTAL);
  hipMallocManaged((void **)&C, sizeof(float) * M_TOTAL * N_TOTAL);
  hipMallocManaged((void **)&D, sizeof(float) * M_TOTAL * N_TOTAL);
  
  // Init matrix A B C on host
  //InitHostMatrix(host_A, host_B, host_C);
  printf("[*] Initializing Matrix...\n");
  InitMatrix(A, B, C, tiles);
  printf("[+]   A: %d x %d\n", M_TOTAL, K_TOTAL);
  printf("[+]   B: %d x %d\n", K_TOTAL, N_TOTAL);
  printf("[+]   C: %d x %d\n", M_TOTAL, N_TOTAL);
  
  // computing gemm using tensor core
  printf("[*] Computing D = A * B +C with Tensor Cores...\n");
  // D = A * B +C, D holds the result after ret
  cuda_status = CalcWMMA(A, B, C, D,tiles);
  
  cuda_status = hipDeviceReset();
  if (cuda_status != hipSuccess) {
    printf("hipDeviceReset failed! ");
    return 1;
  }
  // Todo: Add a function to verify the result by using the result of CPU version implementation.

  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(D);

  return 0;
}