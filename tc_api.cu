//////////////////////////////////////////////////////////////////////
// A simple example to show how CUDA WMMA API works with Tensor Cores
//    Created by Zong-Sheng Wang @ 2018/11/25
// Performance Tips:
//    To minimize bank conflicts, you should try to shift row or 
// column of matrics in shared memory
// cmd: 
//    $ nvcc -o main main.cu -arch sm_75

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <assert.h>


#include "omp.h"
// NVML
#include <nvml.h>

#define WARP_SIZE 32

// MMA matrix tile dimensions.
#define M 8
#define N 8
#define K 4

// // GEMM configuration.
// #define M_TILES 512
// #define N_TILES 512
// #define K_TILES 512
// #define TILE 4
// #define M_TOTAL (M * M_TILES)
// #define N_TOTAL (N * N_TILES)
// #define K_TOTAL (K * K_TILES)
#define ILP 8

//__global__ void WMMAINT8()
using namespace nvcuda;
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
 }                                                                 \
}




__global__ void WMMAF16TensorCore(double *A, double *B,  double *C, int tile_c, int repeat)
{

  wmma::fragment<wmma::matrix_a, M, N, K, double, wmma::row_major> a_frag;
  wmma::fragment<wmma::matrix_b, M, N, K, double, wmma::col_major> b_frag;
  wmma::fragment<wmma::accumulator, M, N, K, double> c_frag[ILP];
  int id_warps=blockIdx.x/32;
  #pragma unroll
  for(int i=0; i<ILP;i++)
  {
    wmma::fill_fragment(c_frag[i], 0.0f);
  }
  
  wmma::load_matrix_sync(a_frag, A , 4);
  wmma::load_matrix_sync(b_frag, B , 4);
  for(int i=0; i<repeat; i++)
  {
    #pragma unroll
    for(int i=0; i<ILP;i++)
    {
      wmma::mma_sync(c_frag[i], a_frag, b_frag, c_frag[i]);
    }
  }
  #pragma unroll
  for(int i=0; i<ILP;i++)
  {
    wmma::store_matrix_sync(C+N*(blockIdx.x+i*gridDim.x) + tile_c*(id_warps), c_frag[i], tile_c, wmma::mem_row_major);
  }
}
#include<unistd.h>   

int main(int argc, char const *argv[])
{
  hipError_t cuda_status;
  int sm_count;
  hipDeviceGetAttribute ( &sm_count, hipDeviceAttributeMultiprocessorCount,0 );
  int num_warps=2;
  int tilesC=sm_count*ILP*N;
  int sizeofmem=sizeof(double)*64;
  int sizeofrmem=sizeofmem*sm_count*ILP*num_warps;

  double *h_mat=(double*)malloc(sizeofmem);
  double *h_mat2=(double*)malloc(sizeofmem);
  double *d_A;
  double *d_B;
  double *d_C;
  hipMalloc((void**)&d_A, sizeofmem);
  hipMalloc((void**)&d_B, sizeofmem);
  hipMalloc((void**)&d_C, sizeofrmem);
  double *h_C=(double*)malloc(sizeofrmem);

  for(int i=0; i<64; i++)
  {
    h_mat[i]=i+1;
  }
  int repeat=100000; 
  int outrepeat=1000;
  hipMemcpy(d_A, h_mat, sizeofmem, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_mat, sizeofmem, hipMemcpyHostToDevice);
  
  float milliseconds = 0;
  nvmlReturn_t result;
  nvmlDevice_t device;
  result=nvmlInit();
  nvmlEnableState_t mode;
  result = nvmlDeviceGetHandleByIndex(0, &device);
  result=nvmlDeviceGetPowerManagementMode(device, &mode);

  #pragma omp parallel num_threads(2)
  {
    
    if (omp_get_thread_num() == 1)
    {
      for(int i=0; i<20;i++)
      {
        unsigned int power1, power2;
        result=nvmlDeviceGetPowerUsage(device,&power1);
        // cuda_status = cudaDeviceSynchronize();
        result=nvmlDeviceGetPowerUsage(device,&power2);

        assert(NVML_SUCCESS == result);
        printf("%d power from %u W to %u W\n", i,
                        power1/1000, power2/1000);
        sleep(1);

      }
      
    }
    if (omp_get_thread_num() == 0){
      hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);
      hipEventRecord(start);
      for(int i=0; i<outrepeat; i++)
      {
        WMMAF16TensorCore<<<sm_count,32*num_warps>>>(d_A, d_B, d_C, tilesC, repeat);
      }
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      
      hipEventElapsedTime(&milliseconds, start, stop);
      printf("[+] GPU(with Tensor Cores) Elapsed Time: %f ms\n", milliseconds);
      printf("[+] TFLOPS: %.2f\n", ((float)8*4*8*2 *ILP*num_warps*sm_count * 2)*repeat*outrepeat / milliseconds / 1e9);
  
    }
  }

  nvmlShutdown();

  
  

  hipMemcpy(h_C, d_C, sizeofrmem, hipMemcpyDeviceToHost);

  return 0;
}
